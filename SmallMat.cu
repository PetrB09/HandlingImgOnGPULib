
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <HandlingImgOnGPU.h>

SmallMat::SmallMat(char* data, int rows, int cols, int x, int y, int sizeOfElement)
{
	Data = data;
	Rows = rows;
	Cols = cols;
	X = x;
	Y = y;
	SizeOfElement = sizeOfElement;
}

int SmallMat::GetRows()
{
	return Rows;
}
int SmallMat::GetCols()
{
	return Cols;
}
char* SmallMat::GetData()
{
	return Data;
}
SmallMat::~SmallMat()
{
	hipFree(Data);
}
