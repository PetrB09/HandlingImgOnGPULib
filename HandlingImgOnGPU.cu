#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <vector>

#include <hip/hip_runtime.h>

//static const int WORK_SIZE = 256;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 *
 * See hip/hip_runtime.h for error code descriptions.
 */
#define CHECK_CUDA_RESULT(N) {											\
	hipError_t result = N;												\
	if (result != 0) {													\
		printf("CUDA call on line %d returned error %d\n", __LINE__,	\
			result);													\
		exit(1);														\
	} }

/*__global__ void cuttngIMG(void* input_ptr, void** output_ptr, int size, int height, int width)
{
	int num = blockIdx.x * blockDim.x + threadIdx.x; //линейный адрес потока
	int numbers = height * width; //число пикселей
	void* ptr_first = input_ptr + num * numbers * size; //указатель на начало фрагмента
	void* output_ptr_2 = &output_ptr + sizeof(void*) * num;
	for(int i = 0; i<height; i++)
	{
		for(int j = 0; j<width; j++)
		{
			int offset = i*blockDim * width + width * blockIdx.x + j;
			*(output_ptr_2 + i*height + j) = *(ptr_first + offset);
		}
	}
}*/

void loadtoGPUmem(void* d_ptr, int size, int height, int width, void* src)
{
	hipMalloc(&d_ptr, size*height*width);
	hipMemcpy(&d_ptr, src, size*height*width, hipMemcpyHostToDevice);
	return;
}

void deleteFromGPUMem(void* d_ptr)
{
	hipFree(d_ptr);
}

void cutImg(void* d_ptr, void* arrOfPointers, int height, int width, int numbersPatchesH, int numbersPatchesW)
{
	if((height % numbersPathchesH) != 0)//1) проверяем делимость размеров на число фрагментов
		return;
	if((width % numbersPatchesW) != 0)
		return;


	int sizeArrOfPointers = numbersPatchesH * numbersPatchesW;
	for(int i = 0; i<sizeArrOfPointers; i++)
	{

	}
	//2) выделяем память, записывая указатели в массив
	//3) запускаем копирование
}

