#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <mem.h>;

#include <hip/hip_runtime.h>

//static const int WORK_SIZE = 256;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 *
 * See hip/hip_runtime.h for error code descriptions.
 */
#define CHECK_CUDA_RESULT(N) {											\
	hipError_t result = N;												\
	if (result != 0) {													\
		printf("CUDA call on line %d returned error %d\n", __LINE__,	\
			result);													\
		exit(1);														\
	} }
/**
 *input_ptr - pointer to sourse array
 *arrOfPtr - pointer to array of pointers to memory for copies
 *size - size of element of sourse array
 *height - height of sourse image
 *width - width of sourse image
 *newHeight - height of new image
 *newWidth - width of new image
 */
__global__ void cuttngIMG(void* input_ptr, void** arrOfPtr, int size, int height, int width, int newHeight, int newWidth)
{
	int NumberThread = blockIdx.x * blockDim.x + threadIdx.x; //linear address of thread
	int SizeNewImg = newHeight * newWidth * size; //size of new picture
	void* SrcElement = input_ptr + blockIdx.x * SizeNewImg + newWidth * size * threadIdx; //input pointer + all blocks which are highly + all elements which are rightly
	void* NewElement = *(arrOfPtr + NumberThread * sizeof(void*));
	for(int i = 0; i<newHeight; i++)
	{
		for(int j = 0; j<newWidth; j++)
		{
			memCpy()
			NewElement += size;
			SrcElement += size;
		}
	}
}

void loadtoGPUmem(void* d_ptr, int size, int height, int width, void* src)
{
	hipMalloc(&d_ptr, size*height*width);
	hipMemcpy(&d_ptr, src, size*height*width, hipMemcpyHostToDevice);
	return;
}

void deleteFromGPUMem(void* d_ptr)
{
	hipFree(d_ptr);
}

void cutImg(void* d_ptr, void** arrOfPointers, int height, int width, int numbersPatchesH, int numbersPatchesW, int size)
{
	if((height % numbersPathchesH) != 0)//1) проверяем делимость размеров на число фрагментов
		return;
	if((width % numbersPatchesW) != 0)
		return;

	int newSizeH = height/numbersPatchesH;
	int newSizeW = width/numbersPatchesW;
	int sizeArrOfPointers = numbersPatchesH * numbersPatchesW;
	for(int i = 0; i<sizeArrOfPointers; i++)//2) выделяем память, записывая указатели в массив
	{
		hipMalloc(&(arrOfPointers + i * sizeof(void*)), newSizeH*newSizeW*size);
	}

	//3) запускаем копирование
}

