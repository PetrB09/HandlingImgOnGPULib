#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <HandlingImgOnGPU.h>

//#include <hip/hip_runtime.h>

//static const int WORK_SIZE = 256;

/**
 *input_ptr - pointer to sourse array
 *arrOfPtr - pointer to array of pointers to memory for copies
 *size - size of element of sourse array
 *height - height of sourse image
 *width - width of sourse image
 *newHeight - height of new image
 *newWidth - width of new image
 */
__global__ void cuttingIMG(BigMat* h_output, char* h_input, int newHeight, int newWidth, int sizeOfElem)
{
	int NumberThread = blockIdx.x * blockDim.x + threadIdx.x; //linear address of thread
	int Offset = NumberThread * newHeight * newWidth * sizeOfElem;
	char* OldArr = &(h_input[Offset]); //pointer to start old array
	SmallMat* Worker = &(h_output->Mats[NumberThread]);
	for(int i = 0; i<newHeight * newWidth; i++)
	{
		for(int j = 0; j<sizeOfElem; j++)
		{
			Worker->Data[i+j] = OldArr[i*Offset + j];
		}
	}
}

void loadtoGPUmem(char** d_ptr, char* src, int height, int width, int sizeOfElement)
{
	hipMalloc(d_ptr, sizeof(char)*height*width*sizeOfElement);
	hipMemcpy(*d_ptr, src, sizeof(char) * width * height, hipMemcpyHostToDevice);
	return;
}

void deleteFromGPUMem(char* d_ptr)
{
	hipFree(d_ptr);
}

void cutImg(char* d_ptr, BigMat** arrOfPatches, int height, int width, int numbersPatchesH, int numbersPatchesW, int sizeOfElement)
{
	if((height % numbersPatchesH) != 0)//1) проверяем делимость размеров на число фрагментов
		return;
	if((width % numbersPatchesW) != 0)
		return;

	int newSizeH = height/numbersPatchesH; //размеры новых картинок
	int newSizeW = width/numbersPatchesW;
	BigMat h_Mats(numbersPatchesH, numbersPatchesW);
	SmallMat* pointer;
	SmallMat** tmp_pointer = &pointer;
	hipMalloc(tmp_pointer, sizeof(SmallMat)* numbersPatchesH * numbersPatchesW); //память под маленькие картинки
	h_Mats.SetData(pointer);
	cuttingIMG<<<numbersPatchesH, numbersPatchesW>>>(&h_Mats, d_ptr, newSizeH, newSizeW, sizeOfElement);
	//запись новых картинок и указателей на них
	*arrOfPatches = &h_Mats;
	return;
}

