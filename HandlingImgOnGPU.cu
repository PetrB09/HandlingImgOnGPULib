#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <HandlingImgOnGPU.h>

//#include <hip/hip_runtime.h>

//static const int WORK_SIZE = 256;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 *
 * See hip/hip_runtime.h for error code descriptions.
 */
#define CHECK_CUDA_RESULT(N) {											\
	hipError_t result = N;												\
	if (result != 0) {													\
		printf("CUDA call on line %d returned error %d\n", __LINE__,	\
			result);													\
		exit(1);														\
	} }
/**
 *input_ptr - pointer to sourse array
 *arrOfPtr - pointer to array of pointers to memory for copies
 *size - size of element of sourse array
 *height - height of sourse image
 *width - width of sourse image
 *newHeight - height of new image
 *newWidth - width of new image
 */
__global__ void cuttingIMG(uchar4*** input_ptr, uchar4*** arrOfPtr, int newHeight, int newWidth)
{
	int NumberThread = blockIdx.x * blockDim.x + threadIdx.x; //linear address of thread

	uchar4** NewArr = arrOfPtr[NumberThread]; //pointer to new array

	for(int i = 0; i<newHeight; i++)
	{
		for(int j = 0; j<newWidth; j++)
		{
			NewArr[i][j] = *(input_ptr) [blockIdx.x * newHeight + i][threadIdx.x * newWidth + j];
		}
	}
}

void loadtoGPUmem(uchar4*** d_ptr, int height, int width, uchar4*** src)
{
	hipMalloc(*d_ptr, sizeof(uchar4*)*height);
	for(int i = 0; i<height; i++)
	{
		hipMalloc(d_ptr[i], sizeof(uchar4) * width);
		hipMemcpy(d_ptr[i], (*src)[i], sizeof(uchar4) * width, hipMemcpyHostToDevice);
	}
	return;
}

void deleteFromGPUMem(uchar4*** d_ptr, int height, int width)
{
	uchar4** Arr = *d_ptr;
	for(int i = 0; i<height; i++)
	{
		hipFree(Arr[i]);
	}
	hipFree(Arr);
	hipFree(d_ptr);
}

void cutImg(uchar4*** d_ptr, uchar4*** arrOfPointers, int height, int width, int numbersPatchesH, int numbersPatchesW)
{
	if((height % numbersPatchesH) != 0)//1) проверяем делимость размеров на число фрагментов
		return;
	if((width % numbersPatchesW) != 0)
		return;

	int newSizeH = height/numbersPatchesH;
	int newSizeW = width/numbersPatchesW;
	int sizeArrOfPointers = numbersPatchesH * numbersPatchesW;
	hipMalloc(&(arrOfPointers), sizeof(uchar4**) * sizeArrOfPointers);
	for(int i = 0; i<sizeArrOfPointers; i++)//2) выделяем память, записывая указатели в массив
	{
		hipMalloc(&(arrOfPointers[i]), sizeof(uchar4*) * newSizeH);
		for(int j = 0; j<newSizeH; j++)
		{
			hipMalloc(&(arrOfPointers[i][j]), sizeof(uchar4)*newSizeW);
		}
	}
	cuttingIMG<<<numbersPatchesH, numbersPatchesW>>>(d_ptr, arrOfPointers, newSizeH, newSizeW);

}
