
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <vector>

#include <hip/hip_runtime.h>

//static const int WORK_SIZE = 256;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 *
 * See cuda.h for error code descriptions.
 */
#define CHECK_CUDA_RESULT(N) {											\
	hipError_t result = N;												\
	if (result != 0) {													\
		printf("CUDA call on line %d returned error %d\n", __LINE__,	\
			result);													\
		exit(1);														\
	} }

void loadtoGPUmem(void** ptr, int size, int height, int width)
{
	hipMalloc(ptr, size*height*width);
	return;
}
void cutImg(void** ptr, void** ptrs[], int size, int height, int width, int result_height, int result_width)
{
	if(((height%result_height) != 0)||((width%result_width) != 0))
		return;
	for(int i = 0; i<height/result_height; i++)
	{
		for(int j = 0; j< width/result_width; j++)
		{
			hipMalloc(&ptrs[i*(height/result_height) + j], size*result_height*result_width);
		}
	}
}
